#include "config.cuh"
#include <iostream>

namespace hpc {
void cudaCheck_(hipError_t ret, const char *file, int line) {
  if (ret != hipSuccess) {
    std::cout << "CUDA error \"" << hipGetErrorString(ret) << " at " << file
              << ":" << line << '\n';
    exit(EXIT_FAILURE);
  }
}
} // namespace hpc
#include "cuda_timer.cuh"

namespace hpc {
cuda_timer::cuda_timer() {
  hipEventCreate(&start_t);
  hipEventCreate(&end_t);
}

cuda_timer::~cuda_timer() {
  hipEventDestroy(end_t);
  hipEventDestroy(start_t);
}

void cuda_timer::start() { hipEventRecord(start_t); }

void cuda_timer::end() { hipEventRecord(end_t); }

double cuda_timer::dur() const {
  hipEventSynchronize(end_t);
  float ms = 0.0f;
  hipEventElapsedTime(&ms, start_t, end_t);
  return ms / 1000.0f;
}
} // namespace hpc
#include "config.cuh"
#include "cuda_timer.cuh"

namespace hpc {
cuda_timer::cuda_timer() : cuda_timer(nullptr) {}

cuda_timer::cuda_timer(std::shared_ptr<stream> timer_stream) {
  this->timer_stream = std::move(timer_stream);
  cudaCheck(hipEventCreate(&start_t));
  cudaCheck(hipEventCreate(&end_t));
}

cuda_timer::~cuda_timer() {
  cudaCheck(hipEventDestroy(end_t));
  cudaCheck(hipEventDestroy(start_t));
}

void cuda_timer::start() {
  hipStream_t stream = timer_stream ? (hipStream_t)*timer_stream : nullptr;
  cudaCheck(hipEventRecord(start_t, stream));
}

void cuda_timer::end() {
  hipStream_t stream = timer_stream ? (hipStream_t)*timer_stream : nullptr;
  cudaCheck(hipEventRecord(end_t, stream));
}

double cuda_timer::dur() const {
  cudaCheck(hipEventSynchronize(end_t));
  float ms = 0.0f;
  cudaCheck(hipEventElapsedTime(&ms, start_t, end_t));
  return ms / 1000.0f;
}
} // namespace hpc
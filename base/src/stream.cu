#include "config.cuh"
#include "stream.cuh"

namespace hpc {
stream::stream() { cudaCheck(hipStreamCreate(&handle)); }
stream::stream(hipStream_t handle) { this->handle = handle; }
stream::~stream() { cudaCheck(hipStreamDestroy(handle)); }

stream::operator hipStream_t &() { return handle; }
stream::operator hipStream_t const &() const { return handle; }
} // namespace hpc
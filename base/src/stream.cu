#include "config.cuh"
#include "stream.cuh"

namespace hpc {
stream::stream() { cudaCheck(hipStreamCreate(&handle)); }
stream::stream(hipStream_t handle) { this->handle = handle; }
stream::~stream() { cudaCheck(hipStreamDestroy(handle)); }

hipStream_t stream::from_ptr(std::shared_ptr<stream> &stream_ptr) {
  return stream_ptr ? (hipStream_t)*stream_ptr : nullptr;
}

stream::operator hipStream_t &() { return handle; }
stream::operator hipStream_t const &() const { return handle; }
} // namespace hpc
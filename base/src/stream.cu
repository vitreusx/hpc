#include "stream.cuh"

namespace hpc {
stream::stream() { hipStreamCreate(&handle); }
stream::~stream() { hipStreamDestroy(handle); }

stream::operator hipStream_t &() { return handle; }
stream::operator hipStream_t const &() const { return handle; }
} // namespace hpc
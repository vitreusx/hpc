#include "hip/hip_runtime.h"
#include <future>
#include <hpc/config.cuh>
#include <hpc/cpu_timer.h>
#include <hpc/cuda_timer.cuh>
#include <hpc/scoped_timer.h>
#include <iostream>
#include <sstream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>

template <typename T> void cpu_stencil(T const *in, T *out, int size, int R) {
  for (int out_idx = 0; out_idx < size; ++out_idx) {
    T res = (T)0;
    for (int in_idx = out_idx - R; in_idx <= out_idx + R; ++in_idx) {
      if (0 <= in_idx && in_idx < size)
        res += in[in_idx];
    }
    out[out_idx] = res;
  }
}

template <typename T>
__global__ void v1_kernel(T const *in, T *out, int size, int R) {
  int out_idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (out_idx >= size)
    return;

  T res = (T)0;
  for (int in_idx = out_idx - R; in_idx <= out_idx + R; ++in_idx) {
    if (0 <= in_idx && in_idx < size)
      res += in[in_idx];
  }
  out[out_idx] = res;
}

template <typename T>
__host__ __device__ T div_round_up(T const &p, T const &q) {
  return (p + q - 1) / q;
}

template <typename T>
void gpu_stencil_v1(T const *in, T *out, int size, int R, int block_size) {
  dim3 block(block_size);
  dim3 grid(div_round_up(size, block_size));
  v1_kernel<T><<<grid, block>>>(in, out, size, R);
}

template <typename T>
__global__ void v2_kernel(T const *in, T *out, int size, int R) {
  extern __shared__ T sm_in[];

  T out_val = 0;
  int sec_off = (int)blockIdx.x * (int)blockDim.x,
      out_val_off = sec_off + (int)threadIdx.x;
  int laneIdx = (int)threadIdx.x % WARP_SIZE;

  int block_R = div_round_up(R, (int)blockDim.x);
  for (int in_sec_ord = -block_R; in_sec_ord <= block_R; ++in_sec_ord) {
    int in_sec_off = sec_off + in_sec_ord * (int)blockDim.x;

    int sm_in_val_off = in_sec_off + (int)threadIdx.x;
    __syncthreads();
    if (0 <= sm_in_val_off && sm_in_val_off < size)
      sm_in[threadIdx.x] = in[sm_in_val_off];
    else
      sm_in[threadIdx.x] = (T)0;
    __syncthreads();

    for (int tile_off = 0; tile_off < blockDim.x; tile_off += WARP_SIZE) {
      auto in_tile_off = in_sec_off + tile_off;

      if (__any_sync(FULL_MASK,
                     (in_tile_off <= out_val_off + R) ||
                         (out_val_off - R <= in_tile_off + WARP_SIZE - 1))) {
        T in_val = sm_in[tile_off + laneIdx], tmp;

        for (int lane = 0; lane < WARP_SIZE; ++lane) {
          tmp = __shfl_sync(FULL_MASK, in_val, lane);
          auto delta = (in_tile_off + lane) - out_val_off;
          if (-R <= delta && delta <= R)
            out_val += tmp;
        }
      }
    }
  }

  if (0 <= out_val_off && out_val_off < size)
    out[out_val_off] = out_val;
}

template <typename T>
void gpu_stencil_v2(T const *in, T *out, int size, int R, int block_size) {
  dim3 block(block_size);
  dim3 grid(div_round_up(size, block_size));
  auto sm = block_size * sizeof(T);
  v2_kernel<<<grid, block, sm>>>(in, out, size, R);
}

template <typename T>
__global__ void v3_kernel(T const *in, T *out, int size, int R) {
  extern __shared__ T sm_in[];

  T out_val = 0;
  int sec_off = (int)blockDim.x * (int)blockIdx.x;
  int out_idx = sec_off + (int)threadIdx.x;

  int block_R = div_round_up(R, (int)blockDim.x);
  for (int block_delta = -block_R; block_delta <= block_R; ++block_delta) {
    int in_sec_off = sec_off + block_delta * (int)blockDim.x;

    int sm_in_idx = in_sec_off + (int)threadIdx.x;
    __syncthreads();
    if (0 <= sm_in_idx && sm_in_idx < size)
      sm_in[threadIdx.x] = in[sm_in_idx];
    else
      sm_in[threadIdx.x] = (T)0;
    __syncthreads();

    int start_idx = max(in_sec_off, out_idx - R) - in_sec_off;
    int end_idx =
        min(in_sec_off + (int)blockDim.x, out_idx + R + 1) - in_sec_off;
    for (int in_idx = start_idx; in_idx < end_idx; ++in_idx)
      out_val += sm_in[in_idx];
  }

  if (0 <= out_idx && out_idx < size)
    out[out_idx] = out_val;
}

template <typename T>
void gpu_stencil_v3(T const *in, T *out, int size, int R, int block_size) {
  dim3 block(block_size);
  dim3 grid(div_round_up(size, block_size));
  auto sm = block_size * sizeof(T);
  v3_kernel<<<grid, block, sm>>>(in, out, size, R);
}

struct random_source {
  thrust::default_random_engine eng;
  thrust::uniform_real_distribution<float> dist;

  random_source() : eng(time(nullptr)) {}

  __host__ __device__ float operator()(int) { return dist(eng); }
};

bool allclose(float a, float b) { return abs(b - a) < 1e-4 * abs(a); }

void allclose(thrust::host_vector<float> const &A,
              thrust::host_vector<float> const &B) {
  if (A.size() != B.size())
    throw std::runtime_error("A and B do not have the same size");

  for (int idx = 0; idx < A.size(); ++idx) {
    if (!allclose(A[idx], B[idx])) {
      std::stringstream error_ss;
      error_ss << "A and B differ at index " << idx << "(" << A[idx]
               << " != " << B[idx] << ")";
      throw std::runtime_error(error_ss.str());
    }
  }
}

int main() {
  using namespace std::chrono_literals;
  int num_reps = 16;
  auto timeout = 5.0s;

  for (auto size : {1'000, 1'000'000, 64'000'000}) {
    thrust::host_vector<float> host_in(size), host_out(size),
        host_dev_out(size);
    thrust::device_vector<float> dev_in(size), dev_out(size);

    thrust::counting_iterator<int> idx_seq;
    thrust::transform(idx_seq, idx_seq + size, host_in.begin(),
                      random_source());

    auto *host_in_ptr = thrust::raw_pointer_cast(host_in.data());
    auto *host_out_ptr = thrust::raw_pointer_cast(host_out.data());
    auto *dev_in_ptr = thrust::raw_pointer_cast(dev_in.data());
    auto *dev_out_ptr = thrust::raw_pointer_cast(dev_out.data());

    for (auto R : {3, 30, 300, 3'000}) {
      bool cpu_all_completed = true;

      for (int rep = 0; rep < num_reps; ++rep) {
        bool completed = false;
        double dur;

        auto fut = std::async([&]() -> void {
          auto timer = hpc::scoped_timer<hpc::cpu_timer>(dur);
          cpu_stencil(host_in_ptr, host_out_ptr, size, R);
          completed = true;
        });

        fut.wait_for(timeout);
        cpu_all_completed &= completed;

        if (!completed) {
          std::cerr << "cpu for size=" << size << ", R=" << R << " timed out\n";
          break;
        } else {
          std::cout << rep << "," << size << "," << R << ",cpu," << dur << '\n';
        }
      }

      auto run_gpu = [&](std::string const &name, auto kernel) -> void {
        hpc::cuda_timer full_timer, kernel_timer;

        for (int rep = 0; rep < num_reps; ++rep) {
          thrust::fill(dev_out.begin(), dev_out.end(), 0.0f);

          full_timer.start();
          {
            thrust::copy(host_in.begin(), host_in.end(), dev_in.begin());
            kernel_timer.start();
            { kernel(); }
            kernel_timer.end();
            thrust::copy(dev_out.begin(), dev_out.end(), host_dev_out.begin());
          }
          full_timer.end();

          std::cout << rep << "," << size << "," << R << "," << name << "-full,"
                    << full_timer.dur() << '\n';
          std::cout << rep << "," << size << "," << R << "," << name
                    << "-kernel," << kernel_timer.dur() << '\n';

          if (cpu_all_completed) {
            allclose(host_out, host_dev_out);
          }
        }
      };

      for (auto block_size : {32, 64, 128, 192, 256, 512, 1024}) {
        auto bs = std::to_string(block_size);
        run_gpu("gpu-v1-" + bs, [&]() -> void {
          gpu_stencil_v1(dev_in_ptr, dev_out_ptr, size, R, block_size);
        });
        run_gpu("gpu-v2-" + bs, [&]() -> void {
          gpu_stencil_v2(dev_in_ptr, dev_out_ptr, size, R, block_size);
        });
        run_gpu("gpu-v3-" + bs, [&]() -> void {
          gpu_stencil_v3(dev_in_ptr, dev_out_ptr, size, R, block_size);
        });
      }
    }
  }
  return EXIT_SUCCESS;
}
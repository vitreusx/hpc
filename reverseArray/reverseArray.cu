
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 768
#define ARRAY_SIZE THREADS_PER_BLOCK * 1024

static void HandleError(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void reverseArray(int *inArray, int *outArray) {
  int in_start = blockDim.x * blockIdx.x;
  int out_start = blockDim.x * (gridDim.x - 1 - blockIdx.x);
  int in_off = threadIdx.x;
  int out_off = blockDim.x - 1 - threadIdx.x;

  outArray[out_start + out_off] = inArray[in_start + in_off];
}

int main(void) {
  int *hostArray;
  int *devInArray, *devOutArray;

  int numBlocks = ARRAY_SIZE / THREADS_PER_BLOCK;

  size_t memSize = ARRAY_SIZE * sizeof(int);
  hostArray = (int *)malloc(memSize);
  HANDLE_ERROR(hipMalloc((void **)&devInArray, memSize));
  HANDLE_ERROR(hipMalloc((void **)&devOutArray, memSize));

  for (int i = 0; i < ARRAY_SIZE; i++) {
    hostArray[i] = i;
  }

  HANDLE_ERROR(
      hipMemcpy(devInArray, hostArray, memSize, hipMemcpyHostToDevice));

  dim3 dimGrid(numBlocks);
  dim3 dimBlock(THREADS_PER_BLOCK);
  reverseArray<<<dimGrid, dimBlock>>>(devInArray, devOutArray);

  hipDeviceSynchronize();

  HANDLE_ERROR(
      hipMemcpy(hostArray, devOutArray, memSize, hipMemcpyDeviceToHost));

  for (int i = 0; i < ARRAY_SIZE; i++) {
    assert(hostArray[i] == ARRAY_SIZE - 1 - i);
  }

  HANDLE_ERROR(hipFree(devInArray));
  HANDLE_ERROR(hipFree(devOutArray));

  free(hostArray);

  printf("Correct!\n");
  return 0;
}

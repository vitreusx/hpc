#include "tex_bitmap.cuh"

tex_bitmap::tex_bitmap(int w, int h) {
  this->w = w;
  this->h = h;

  auto channel_desc = hipCreateChannelDesc<unsigned char>();
  hipMallocArray(&pixels, &channel_desc, w, h);

  hipResourceDesc res_desc = {};
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = pixels;

  hipTextureDesc tex_desc = {};
  tex_desc.addressMode[0] = hipAddressModeWrap;
  tex_desc.addressMode[1] = hipAddressModeWrap;
  tex_desc.filterMode = hipFilterModeLinear;
  tex_desc.readMode = hipReadModeElementType;
  tex_desc.normalizedCoords = 1;

  tex = {};
  hipCreateTextureObject(&tex, &res_desc, &tex_desc, nullptr);
}

tex_bitmap::~tex_bitmap() {
  hipDestroyTextureObject(tex);
  hipFree(pixels);
}

tex_bitmap::operator tex_bitmap_view() {
  tex_bitmap_view view;
  view.tex = tex;
  view.w = w;
  view.h = h;
  return view;
}